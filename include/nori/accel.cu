/*
    This file is part of Nori, a simple educational ray tracer

    Copyright (c) 2015 by Wenzel Jakob

    Nori is free software; you can redistribute it and/or modify
    it under the terms of the GNU General Public License Version 3
    as published by the Free Software Foundation.

    Nori is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program. If not, see <http://www.gnu.org/licenses/>.
*/

#pragma once

#include <nori/mesh.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


NORI_NAMESPACE_BEGIN



struct BVHmesh
{
    std::vector<Mesh *> originalMeshes;
	size_t nMeshes = 0;
	MatrixXf      m_V;                   ///< Vertex positions
	MatrixXu      m_F;                   ///< Faces
	BoundingBox3f m_bbox;                ///< Bounding box

	const Mesh *getOriginalMesh(n_UINT index) const { return originalMeshes[index]; }

	void uploadToDevice(MatrixXf *&d_V, MatrixXu *&d_F) const
	{
		hipMalloc(&d_V, m_V.size() * sizeof(float));
		hipMalloc(&d_F, m_F.size() * sizeof(n_UINT));

		hipMemcpy(d_V, m_V.data(), m_V.size() * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_F, m_F.data(), m_F.size() * sizeof(n_UINT), hipMemcpyHostToDevice);
	}

	void clear() {
		m_V.resize(3, 0);
		m_F.resize(3, 0);
		originalMeshes.clear();
		m_bbox = BoundingBox3f();
	}

	__host__ __device__
	bool rayIntersect(n_UINT index, 
			const Ray3f &ray, float &u, float &v, float &t) const
	{
		n_UINT i0 = m_F(0, index), i1 = m_F(1, index), i2 = m_F(2, index);
    	const Eigen::Vector3f p0 = m_V.col(i0), p1 = m_V.col(i1), p2 = m_V.col(i2);

		/* Find vectors for two edges sharing v[0] */
		Eigen::Vector3f edge1 = p1 - p0, edge2 = p2 - p0;

		/* Begin calculating determinant - also used to calculate U parameter */
		Eigen::Vector3f pvec = ray.d.cross(edge2);

		/* If determinant is near zero, ray lies in plane of triangle */
		float det = edge1.dot(pvec);

		if (det > -1e-8f && det < 1e-8f)
			return false;
		float inv_det = 1.0f / det;

		/* Calculate distance from v[0] to ray origin */
		Eigen::Vector3f tvec = ray.o - p0;

		/* Calculate U parameter and test bounds */
		u = tvec.dot(pvec) * inv_det;
		if (u < 0.0 || u > 1.0)
			return false;

		/* Prepare to test V parameter */
		Eigen::Vector3f qvec = tvec.cross(edge1);

		/* Calculate V parameter and test bounds */
		v = ray.d.dot(qvec) * inv_det;
		if (v < 0.0 || u + v > 1.0)
			return false;

		/* Ray intersects triangle -> compute t */
		t = edge2.dot(qvec) * inv_det;

		return t >= ray.mint && t <= ray.maxt;
	}

	BoundingBox3f getBoundingBox(n_UINT index) const {
		BoundingBox3f result(m_V.col(m_F(0, index)));
		result.expandBy(m_V.col(m_F(1, index)));
		result.expandBy(m_V.col(m_F(2, index)));
		return result;
	}

	Point3f getCentroid(n_UINT index) const {
		return (1.0f / 3.0f) *
			(m_V.col(m_F(0, index)) +
			m_V.col(m_F(1, index)) +
			m_V.col(m_F(2, index)));
	}

	size_t size() const { return m_F.cols(); }


	void append(Mesh *other, size_t meshId)
	{
		if (m_V.cols() == 0)
		{
			m_V = other->getVertexPositions();
			m_F = other->getIndices();
		}
		else
		{
			m_V.conservativeResize(3, m_V.cols() + other->getVertexPositions().cols());
			m_F.conservativeResize(3, m_F.cols() + other->getIndices().cols());

			m_V.rightCols(other->getVertexPositions().cols()) = other->getVertexPositions();
			m_F.rightCols(other->getIndices().cols()) = other->getIndices();
		}

		// Add the original mesh
		for (size_t i = 0; i < other->getIndices().cols(); ++i)
			originalMeshes.push_back(other);

		m_bbox.expandBy(other->getBoundingBox());
	}
};  

/**
 * \brief Acceleration data structure for ray intersection queries
 *
 * The current implementation falls back to a brute force loop
 * through the geometry.
 */
class Accel {
	friend class BVHBuildTask;
public:
	/// Create a new and empty BVH
	Accel() { m_meshOffset.push_back(0u); }

	/// Release all resources
	void clear();

	/// Release all resources
	virtual ~Accel() { clear(); };

	/**
	 * \brief Register a triangle mesh for inclusion in the BVH.
	 *
	 * This function can only be used before \ref build() is called
	 */
	void addMesh(Mesh *mesh);

	/// Build the BVH
	void build();

	/**
	 * \brief Intersect a ray against all triangle meshes registered
	 * with the BVH
	 *
	 * Detailed information about the intersection, if any, will be
	 * stored in the provided \ref Intersection data record.
	 *
	 * The <tt>shadowRay</tt> parameter specifies whether this detailed
	 * information is really needed. When set to \c true, the
	 * function just checks whether or not there is occlusion, but without
	 * providing any more detail (i.e. \c its will not be filled with
	 * contents). This is usually much faster.
	 *
	 * \return \c true If an intersection was found
	 */
	__host__ __device__
	bool rayIntersect(const Ray3f &ray, Intersection &its,
		bool shadowRay = false) const;

	void rayIntersect(const std::vector<bool> &mask, 
						const std::vector<Ray3f> &ray, 
						std::vector<Intersection> &its,
						std::vector<bool> &b_its) const;

	bool rayProbe(const Ray3f &_ray, 
		std::vector<Intersection> &its) const;

	/// Return the total number of meshes registered with the BVH
	n_UINT getMeshCount() const { return (n_UINT)m_meshes.size(); }

	/// Return the total number of internally represented triangles 
	n_UINT getTriangleCount() const { return globalMesh.size(); }

	/// Return one of the registered meshes
	Mesh *getMesh(n_UINT idx) { return m_meshes[idx]; }

	/// Return one of the registered meshes (const version)
	const Mesh *getMesh(n_UINT idx) const { return m_meshes[idx]; }

	//// Return an axis-aligned bounding box containing the entire tree
	const BoundingBox3f &getBoundingBox() const {
		return m_bbox;
	}

public:
	/**
	 * \brief Compute the mesh and triangle indices corresponding to
	 * a primitive index used by the underlying generic BVH implementation.
	 */
	n_UINT findMesh(n_UINT &idx) const {
		auto it = std::lower_bound(m_meshOffset.begin(), m_meshOffset.end(), idx + 1) - 1;
		idx -= *it;
		return (n_UINT)(it - m_meshOffset.begin());
	}

	//// Return an axis-aligned bounding box containing the given triangle
	BoundingBox3f getBoundingBox(n_UINT index) const {
		return globalMesh.getBoundingBox(index);
	}

	//// Return the centroid of the given triangle
	Point3f getCentroid(n_UINT index) const {
		return globalMesh.getCentroid(index);
	}

	void compactMeshes();

	/// Compute internal tree statistics
	std::pair<float, n_UINT> statistics(n_UINT index = 0) const;

	Intersection fillIntersection(Point2f uv, n_UINT f, float t) const;

	/* BVH node in 32 bytes */
	struct BVHNode {
		union {
			struct {
				unsigned flag : 1;
				uint32_t size : 31;
				n_UINT start;
			} leaf;

			struct {
				unsigned flag : 1;
				uint32_t axis : 31;
				n_UINT rightChild;
			} inner;

			uint64_t data;
		};
		BoundingBox3f bbox;

		__host__ __device__
		bool isLeaf() const {
			return leaf.flag == 1;
		}

		__host__ __device__
		bool isInner() const {
			return leaf.flag == 0;
		}

		__host__ __device__
		bool isUnused() const {
			return data == 0;
		}

		__host__ __device__
		n_UINT start() const {
			return leaf.start;
		}

		__host__ __device__
		n_UINT end() const {
			return leaf.start + leaf.size;
		}
	};
private:
	std::vector<Mesh *> m_meshes;       ///< List of meshes registered with the BVH
	std::vector<n_UINT> m_meshOffset; ///< Index of the first triangle for each shape
	thrust::host_vector<n_UINT> m_indices;    ///< Index references by BVH nodes
	thrust::device_vector<n_UINT> device_indices;    ///< Index references by BVH nodes


	thrust::host_vector<BVHNode> m_nodes;       ///< BVH nodes
	thrust::device_vector<BVHNode> device_nodes; ///< BVH nodes

	BVHmesh globalMesh;                 ///< Global mesh containing all triangles
	MatrixXf *deviceVertices;            ///< Global vertex positions
	MatrixXu *deviceFaces;             ///< Global triangle indices

	BoundingBox3f m_bbox;               ///< Bounding box of the entire BVH
};


NORI_NAMESPACE_END
